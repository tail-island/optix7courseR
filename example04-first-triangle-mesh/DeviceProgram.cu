#include "hip/hip_runtime.h"
#include <tuple>

#pragma nv_diag_suppress 20236

#include <Eigen/Core>
#include <Eigen/Geometry>
#include <optix.h>
#include <optix_device.h>
#include <optix_stubs.h>

#include "OptixParams.h"

namespace osc {

extern "C" {
__constant__ LaunchParams optixLaunchParams;
}

// OptiXのペイロードはunsigned int×n個で扱いづらいので、構造体へのポインタに変換します。

inline __device__ auto getPayloadParams(void *payloadPointer) noexcept {
  auto p = reinterpret_cast<std::uint64_t>(payloadPointer);

  return std::make_tuple(static_cast<std::uint32_t>(p >> 32), static_cast<std::uint32_t>(p));
}

inline __device__ auto getPayloadPointer() noexcept {
  return reinterpret_cast<void *>(static_cast<std::uint64_t>(optixGetPayload_0()) << 32 | static_cast<std::uint64_t>(optixGetPayload_1()));
}

// 色は、とりあえずポリゴンのインデックスから適当に作ります。

inline __device__ auto getRandomColor(unsigned int seed) noexcept {
  const auto r = seed * 13 * 17 + 0x234235;
  const auto g = seed * 7 * 3 * 5 + 0x773477;
  const auto b = seed * 11 * 19 + 0x223766;

  return Eigen::Vector3f((r & 0x00ff) / 255.0f, (g & 0x00ff) / 255.0f, (b & 0x00ff) / 255.0f);
}

// 光を生成します。

extern "C" __global__ void __raygen__renderFrame() {
  const auto &x = optixGetLaunchIndex().x;
  const auto &y = optixGetLaunchIndex().y;

  const auto &camera = reinterpret_cast<RaygenData *>(optixGetSbtDataPointer())->camera;

  auto origin = camera.origin;
  auto direction = ((static_cast<float>(x) / optixGetLaunchDimensions().x * 2 - 1) * camera.u + (static_cast<float>(y) / optixGetLaunchDimensions().y * 2 - 1) * camera.v + camera.w).normalized();

  auto color = Eigen::Vector3f{0};

  auto [payloadParam0, payloadParam1] = getPayloadParams(&color);

  optixTrace(
      optixLaunchParams.traversableHandle,
      *reinterpret_cast<float3 *>(&origin),
      *reinterpret_cast<float3 *>(&direction),
      0.0f,                               // tmin
      1e20f,                              // tmax
      0.0f,                               // rayTime
      OptixVisibilityMask(255),           //
      OPTIX_RAY_FLAG_DISABLE_ANYHIT,      // rayFlags,
      static_cast<int>(RayType::Surface), // SBToffset
      static_cast<int>(RayType::Size),    // SBTstride
      static_cast<int>(RayType::Surface), // missSBTIndex
      payloadParam0,                      // ペイロードではunsigned intしか使えません……。
      payloadParam1);

  const auto r = static_cast<int>(255.5 * color.x()); // intへのキャストは小数点以下切り捨てなので、255よりも少し大きい値を使用しました。
  const auto g = static_cast<int>(255.5 * color.y());
  const auto b = static_cast<int>(255.5 * color.z());

  optixLaunchParams.imageBuffer[x + y * optixGetLaunchDimensions().x] = r << 0 | g << 8 | b << 16 | 0xff000000;
}

// 物体に光が衝突した場合の処理です。衝突判定は自動でやってくれるみたい。

extern "C" __global__ void __closesthit__radiance() {
  *reinterpret_cast<Eigen::Vector3f *>(getPayloadPointer()) = getRandomColor(optixGetPrimitiveIndex()); // とりあえず、光が衝突したポリゴンのインデックスをシードにして、ランダムな色を割り当てます。
}

// 物体に光が衝突しそうな場合の処理？

extern "C" __global__ void __anyhit__radiance() {
  ; // とりあえず、なにもしません。
}

// トレースした光が物体に衝突しなかった場合の処理です。

extern "C" __global__ void __miss__radiance() {
  *reinterpret_cast<Eigen::Vector3f *>(getPayloadPointer()) = Eigen::Vector3f{1, 1, 1}; // とりあえず、背景は真っ白にします。
}

} // namespace osc
