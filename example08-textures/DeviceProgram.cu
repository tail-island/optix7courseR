#include "hip/hip_runtime.h"
#include <tuple>

#pragma nv_diag_suppress 20236

#include <Eigen/Core>
#include <Eigen/Geometry>
#include <optix.h>
#include <optix_device.h>
#include <optix_stubs.h>

#include "OptixParams.h"

namespace osc {

extern "C" {
__constant__ LaunchParams optixLaunchParams;
}

struct HitgroupData {
  TriangleMeshes triangleMeshes;
};

// OptiXのペイロードはunsigned int×n個で扱いづらいので、構造体へのポインタに変換します。

inline __device__ auto getPayloadParams(void *payloadPointer) noexcept {
  auto p = reinterpret_cast<std::uint64_t>(payloadPointer);

  return std::make_tuple(static_cast<std::uint32_t>(p >> 32), static_cast<std::uint32_t>(p));
}

inline __device__ auto getPayloadPointer() noexcept {
  return reinterpret_cast<void *>(static_cast<std::uint64_t>(optixGetPayload_0()) << 32 | static_cast<std::uint64_t>(optixGetPayload_1()));
}

// レイを生成します。

extern "C" __global__ void __raygen__renderFrame() {
  const auto &x = optixGetLaunchIndex().x;
  const auto &y = optixGetLaunchIndex().y;

  // カメラの情報を取得します。

  auto &origin = *reinterpret_cast<Eigen::Vector3f *>(&optixLaunchParams.camera.origin); // optixTraceの都合で、const autoに出来ない……。

  const auto &u = *reinterpret_cast<Eigen::Vector3f *>(&optixLaunchParams.camera.u);
  const auto &v = *reinterpret_cast<Eigen::Vector3f *>(&optixLaunchParams.camera.v);
  const auto &w = *reinterpret_cast<Eigen::Vector3f *>(&optixLaunchParams.camera.w);

  // レイの方向を計算します。

  auto direction = ((static_cast<float>(x) / optixGetLaunchDimensions().x * 2 - 1) * u + (static_cast<float>(y) / optixGetLaunchDimensions().y * 2 - 1) * v + w).normalized(); // optixTraceの都合で、const autoに出来ない……。

  // ピクセルの色を表現する変数を用意します。この値をoptixTraceして設定します。

  auto color = Eigen::Vector3f{0};
  auto [payloadParam0, payloadParam1] = getPayloadParams(&color); // optixTraceの都合で、const autoに出来ない……。

  // optixTraceして、レイをトレースします。

  optixTrace(
      optixLaunchParams.traversableHandle,
      *reinterpret_cast<float3 *>(&origin),
      *reinterpret_cast<float3 *>(&direction),
      0.0f,                                // tmin
      1e20f,                               // tmax
      0.0f,                                // rayTime
      OptixVisibilityMask(255),            //
      OPTIX_RAY_FLAG_DISABLE_ANYHIT,       // rayFlags,
      static_cast<int>(RayType::Radiance), // SBToffset
      static_cast<int>(RayType::Size),     // SBTstride
      static_cast<int>(RayType::Radiance), // missSBTIndex
      payloadParam0,                       // ペイロードではunsigned intしか使えません……。
      payloadParam1);

  // optixTraceで設定されたcolorを使用して、イメージ・バッファーに値を設定します。

  optixLaunchParams.imageBuffer[x + y * optixGetLaunchDimensions().x] = float3{color.x(), color.y(), color.z()};
}

// 物体にレイが衝突した場合の処理です。衝突判定は自動でやってくれます。

extern "C" __global__ void __closesthit__radiance() {
  const auto &triangleMeshes = reinterpret_cast<HitgroupData *>(optixGetSbtDataPointer())->triangleMeshes;

  const auto &index = triangleMeshes.indices[optixGetPrimitiveIndex()];

  const auto u = optixGetTriangleBarycentrics().x;
  const auto v = optixGetTriangleBarycentrics().y;

  // レイが衝突した場所の色を取得します。

  const auto color = [&] {
    if (!triangleMeshes.hasTextureObject) {
      return triangleMeshes.color;
    }

    const auto textureCoordinate = (1 - u - v) * triangleMeshes.textureCoordinates[index.x()] + u * triangleMeshes.textureCoordinates[index.y()] + v * triangleMeshes.textureCoordinates[index.z()];
    const auto textureColor = tex2D<float4>(triangleMeshes.textureObject, textureCoordinate.x(), textureCoordinate.y());

    return Eigen::Vector3f{textureColor.x, textureColor.y, textureColor.z};
  }();

  // レイが衝突した場所の法線を取得します。

  const auto normal = [&] {
    return ((1 - u - v) * triangleMeshes.normals[index.x()] + u * triangleMeshes.normals[index.y()] + v * triangleMeshes.normals[index.z()]).normalized();
  }();

  // レイの向きを取得します。

  const auto rayDirection = [&] {
    auto result = optixGetWorldRayDirection();

    return *reinterpret_cast<Eigen::Vector3f *>(&result);
  }();

  // 色は、光源とかはとりあえず考慮しないで、レイとポリゴンが垂直なほど明るくなるということで。カメラにライトが付いているとでも思って、納得してください……。

  *reinterpret_cast<Eigen::Vector3f *>(getPayloadPointer()) = color * (0.2 + 0.8 * std::abs(normal.dot(rayDirection)));
}

// 物体にレイが衝突しそうな場合の処理です。このコースでは最後まで使用しません。

extern "C" __global__ void __anyhit__radiance() {
  ; // このコースでは、なにもしません。
}

// レイが物体に衝突しなかった場合の処理です。

extern "C" __global__ void __miss__radiance() {
  *reinterpret_cast<Eigen::Vector3f *>(getPayloadPointer()) = Eigen::Vector3f{1, 1, 1}; // とりあえず、背景は真っ白にします。
}

} // namespace osc
